#include "hip/hip_runtime.h"

#ifndef PARTICLES_CU
#define PARTICLES_CU

// includes, project
#include <hip/hip_runtime_api.h>

#include "Particles_kernel.cu"
#include "Integration_kernel.cu"


extern "C"
{
	void setATSParameters( ATSParams *hostParams)
	{
		// copy parameters to constant memory
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( params), hostParams, sizeof(ATSParams)) );
	}
	
	void setMaxParticleSteps( int num_steps )
	{
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL( max_particle_steps), &num_steps, sizeof(int),0));
	}


	void AdvectParticlesCuda( float4* dPositions, float4* dVelocities, int numParticles, float timestep, uchar* occupiedBlocks, hipStream_t stream)
    {
		if( numParticles < 1 ) return;

		// setup execution parameters
        int threadsPerBlock = 256; // 256 threads per block
        int numBlocks = (numParticles + threadsPerBlock-1) / threadsPerBlock;

        CREATE_TIMER
        START_TIMER(stream)

        //// execute the kernel
        AdvectParticlesKernel<<< numBlocks, threadsPerBlock, 0, stream >>> ( dPositions, dVelocities, timestep, numParticles, occupiedBlocks );
        getLastCudaError("Kernel execution failed");

        STOP_TIMER(stream)
        PRINT_TIMER("Particle Advection: ", "\n")
        STORE_TIMER(advectionTimer)
        DESTROY_TIMER
    }; 

	void IntegrateVelocitiesCuda
	(
		IntegrationScheme integration,
		float4* dPositions, float4* dVelocities, int numParticles,
		vistime_t* vistime,
		MeshGPU mesh1, MeshGPU mesh2,
		int* dStartCells1, int* dStartCells2, 
		float* dStepSizes, hipStream_t stream
	)
    {

		if( numParticles < 1 ) return;

		checkCudaErrors(hipBindTexture(0, nodes1Tex,		mesh1.nodes,		mesh1.s_nodes));
		checkCudaErrors(hipBindTexture(0, cells1Tex,		mesh1.cells,		mesh1.s_cells));
		checkCudaErrors(hipBindTexture(0, neighbors1Tex,	mesh1.neighbors,	mesh1.s_cells));
		checkCudaErrors(hipBindTexture(0, nodeAttributes1Tex, mesh1.nodeAttributes, mesh1.s_nodes));

		checkCudaErrors(hipBindTexture(0, nodes2Tex,		mesh2.nodes,		mesh2.s_nodes));
		checkCudaErrors(hipBindTexture(0, cells2Tex,		mesh2.cells,		mesh2.s_cells));
		checkCudaErrors(hipBindTexture(0, neighbors2Tex,	mesh2.neighbors,	mesh2.s_cells));
		checkCudaErrors(hipBindTexture(0, nodeAttributes2Tex, mesh2.nodeAttributes, mesh2.s_nodes));

		// copy vistime to constant device memory
		checkCudaErrors( hipMemcpyToSymbolAsync(HIP_SYMBOL( vt),	vistime, sizeof(vistime_t), 0, hipMemcpyHostToDevice, stream ));

		int* nc1;
		allocatePageLockedArray((void**)&nc1, sizeof(int), false);
		*(nc1) = mesh1.num_cells;
		checkCudaErrors( hipMemcpyToSymbolAsync(HIP_SYMBOL( num_cells_m1), nc1, sizeof(int), 0, hipMemcpyHostToDevice, stream));

		int* nc2;
		allocatePageLockedArray((void**)&nc2, sizeof(int), false);
		*(nc2) = mesh2.num_cells;
		checkCudaErrors( hipMemcpyToSymbolAsync(HIP_SYMBOL( num_cells_m2), nc2, sizeof(int), 0, hipMemcpyHostToDevice, stream));

		// reset number of hops for current iteration
		int* dSymbol;
		checkCudaErrors( hipGetSymbolAddress((void**)&dSymbol, num_hops1));
		checkCudaErrors( hipMemset( dSymbol, 0, numParticles * sizeof(int)));
		checkCudaErrors( hipGetSymbolAddress((void**)&dSymbol, num_hops2));
		checkCudaErrors( hipMemset( dSymbol, 0, numParticles * sizeof(int)));

		// reset iterations counter
		checkCudaErrors( hipGetSymbolAddress((void**)&dSymbol, iterations));
		checkCudaErrors( hipMemset( dSymbol, 0, numParticles * sizeof(int)));

		// setup execution parameters
		int threadsPerBlock = 128; // number of Particles per block
		int numBlocks = ( numParticles + threadsPerBlock-1) / threadsPerBlock;

        CREATE_TIMER
        START_TIMER(stream)

		// execute the kernel
		switch( integration )
		{
		default:
		case Euler:
			IntegrateVelocitiesEulerKernel<<< numBlocks, threadsPerBlock, 0, stream >>>
			(
				dPositions, dVelocities, numParticles,
				mesh1.traversedCells, mesh2.traversedCells,
				dStartCells1, dStartCells2
			);
			break;
		case RK3:
			IntegrateVelocitiesRK3Kernel<<< numBlocks, threadsPerBlock, 0, stream >>>
			(
				dPositions, dVelocities, numParticles,
				mesh1.traversedCells, mesh2.traversedCells,
				dStartCells1, dStartCells2
			);
			break;
		case RK4:
			IntegrateVelocitiesRK4Kernel<<< numBlocks, threadsPerBlock, 0, stream >>>
			(
				dPositions, dVelocities, numParticles,
				mesh1.traversedCells, mesh2.traversedCells,
				dStartCells1, dStartCells2
			);
			break;
		case Dopri5:
			IntegrateVelocitiesDopri5Kernel<<< numBlocks, threadsPerBlock, 0, stream >>>
			(
				dPositions, dVelocities, numParticles,
				mesh1.traversedCells, mesh2.traversedCells, 
				dStartCells1, dStartCells2
			);
			break;
		case Dopri5_ATS:
			IntegrateVelocitiesDopri5_ATS_Kernel<<< numBlocks, threadsPerBlock, 0, stream >>>
			(
				dPositions, dVelocities, numParticles,
				mesh1.traversedCells, mesh2.traversedCells,
				dStartCells1, dStartCells2,
				dStepSizes
			);
			break;
		}
   
		getLastCudaError("Kernel execution failed");

        STOP_TIMER(stream)
		PRINT_TIMER("Integration: ", "\n")
		STORE_TIMER(integrationTimer)
        DESTROY_TIMER


		checkCudaErrors(hipUnbindTexture(nodes1Tex));
		checkCudaErrors(hipUnbindTexture(nodeAttributes1Tex));
		checkCudaErrors(hipUnbindTexture(cells1Tex));
		checkCudaErrors(hipUnbindTexture(neighbors1Tex));

		checkCudaErrors(hipUnbindTexture(nodes2Tex));
		checkCudaErrors(hipUnbindTexture(nodeAttributes2Tex));
		checkCudaErrors(hipUnbindTexture(cells2Tex));
		checkCudaErrors(hipUnbindTexture(neighbors2Tex));

		freePageLockedHostMemory(nc1);
		freePageLockedHostMemory(nc2);
	};

	void getNumberOfHops( int count, int* hNumHops1, int* hNumHops2, int* hIterations )
	{
		hipMemcpyFromSymbol(hNumHops1, HIP_SYMBOL(num_hops1), count * sizeof(int), 0, hipMemcpyDeviceToHost );
		hipMemcpyFromSymbol(hNumHops2, HIP_SYMBOL(num_hops2), count * sizeof(int), 0, hipMemcpyDeviceToHost );
		hipMemcpyFromSymbol(hIterations, HIP_SYMBOL(iterations), count * sizeof(int), 0, hipMemcpyDeviceToHost );
	}
}

#endif
