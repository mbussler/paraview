
#ifndef COMMON_CU
#define COMMON_CU

// includes
#include "common.cuh"

extern "C"
{
    void cudaInit(int argc, char **argv)
    {   
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        if( checkCmdLineFlag(argc, (const char**)argv, "device") ) {
            cudaDeviceInit(argc, argv);
        } else {
            hipSetDevice( getMaxGflopsDeviceId() );
        }
    };

    void cudaGLInit(int argc, char **argv)
    {   
        // use command-line specified CUDA device, otherwise use device with highest Gflops/s
        if( checkCmdLineFlag(argc, (const char**)argv, "device") ) {
            cudaDeviceInit(argc, argv);
        } else {
            cudaGLSetGLDevice( getMaxGflopsDeviceId() );
        }
    };

    void allocateArray(void **devPtr, size_t size)
    {
		hipError_t err = hipMalloc(devPtr, size);
	    if( hipSuccess != err) {
			printf((stderr, "%s(%i) : cudaSafeCall() Runtime API error : %s.\n",
					__FILE__, __LINE__, hipGetErrorString( err) ));
			exit(-1);
	    }
    };


    void freeArray(void *devPtr)
    {
		hipError_t err = hipFree(devPtr);
	    if( hipSuccess != err) {
			printf((stderr, "%s(%i) : cudaSafeCall() Runtime API error : %s.\n",
					__FILE__, __LINE__, hipGetErrorString( err) ));
			exit(-1);
	    }
    };

    void allocatePageLockedArray(void **hostPtr, size_t size, bool wc)
    {
#if CUDART_VERSION >= 2020
		checkCudaErrors( hipHostAlloc( hostPtr, size, (wc) ? hipHostMallocWriteCombined : 0));
#else
		cutilSafeCall( hipHostAlloc( hostPtr, size );
#endif
    };

    void allocatePageLockedArrayPortable(void **hostPtr, size_t size, bool wc)
    {
		//cutilSafeCall( hipHostAlloc( hostPtr, size, hipHostMallocPortable | (wc) ? hipHostMallocWriteCombined : 0));
		checkCudaErrors( hipHostAlloc( hostPtr, size, (wc) ? hipHostMallocWriteCombined : 0));
    };

    void freePageLockedHostMemory(void *hostPtr)
    {
		checkCudaErrors( hipHostFree( hostPtr));
    };

    void createStreams( int numStreams, hipStream_t* streams )
    {
    	for (int i = 0; i < numStreams; ++i)
    	{
    	    hipStreamCreate(&streams[i]);
    	}
    }


    void copyArrayToPageLockedHostMemory(void *hostPtr, void* src, size_t size)
    {
		checkCudaErrors( hipMemcpy( hostPtr, src, size, hipMemcpyHostToHost ));
    };

    void setArray(void* devPtr, int value, size_t count)
    {
		checkCudaErrors( hipMemset( devPtr, value, count));
    };

	void copyArrayToDevice(void* device, const void* host, int size)
    {
        checkCudaErrors(hipMemcpy(device, host, size, hipMemcpyHostToDevice) );
    };

	void copyArrayToDeviceAsync(void* device, const void* host, int size, hipStream_t stream)
    {
        checkCudaErrors(hipMemcpyAsync(device, host, size, hipMemcpyHostToDevice, stream));
    };

    void copyArrayFromDevice(void* host, const void* device, int size, struct hipGraphicsResource **cuda_vbo_resource)
    {   

	#ifndef NOGL
		if (cuda_vbo_resource)
			device = mapGLBufferObject(cuda_vbo_resource);
	#endif

        checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
        
	#ifndef NOGL
		if (cuda_vbo_resource)
			unmapGLBufferObject(*cuda_vbo_resource);
	#endif
    };

    void copyArrayFromDeviceAsync(void* host, const void* device, int size, hipStream_t stream)
    {
        checkCudaErrors(hipMemcpyAsync(host, device, size, hipMemcpyDeviceToHost, stream));
    };


    void copyToConstantMem( const char* symbol, const void* src, size_t count )
    {   
        checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL( symbol), src, count, 0, hipMemcpyHostToDevice ));
    };

    void copyToSymbolAsync( const char *symbol, const void *src, size_t count, hipStream_t stream)
    {
    	checkCudaErrors( hipMemcpyToSymbolAsync(HIP_SYMBOL( symbol), src, count, 0, hipMemcpyHostToDevice, stream ));
    }

    void resetSymbol( const char *symbol, size_t count )
    {
		int* dSymbol;
    	checkCudaErrors( hipGetSymbolAddress((void**)&dSymbol, symbol ));
		checkCudaErrors( hipMemset( dSymbol, 0, count ));
    }

    void threadSync()
	{
        checkCudaErrors(hipDeviceSynchronize());
    };

    void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
    {
	#ifndef NOGL
		checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo, cudaGraphicsMapFlagsNone));
	#endif
    };

    void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
	#ifndef NOGL
        checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
	#endif
    };

    void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
    {
        void *ptr = 0;
        
	#ifndef NOGL
		checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
        size_t num_bytes; 
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes, *cuda_vbo_resource));
	#endif
    
		return ptr;
    };

    void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
    {
	#ifndef NOGL
       checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	#endif
    };

	void getGPUMemoryUsage(size_t* free, size_t* total, int divBy)
	{
		checkCudaErrors( hipMemGetInfo( free, total ));
		*free /= divBy;
		*total /= divBy;
	};
 
	//void getTimerMedian(vector<float>* timer, float* med_time )
	//{
	//	if( !timer || !med_time || timer->empty()) return;

	//	vector<float>::iterator midpoint;
	//	midpoint = timer->begin() + (timer->end() - timer->begin())/2;
	//	nth_element(timer->begin(), midpoint, timer->end());
	//	*med_time = *midpoint;
	//};

	//void getTimerAveraged(vector<float>* timer, float* avg_time )
	//{
	//	if( !timer || !avg_time || timer->empty()) return;

	//    float total = 0.0;
	//    for (int i=0; i< timer->size(); i++) {
	//        total += timer->at(i);
	//    }
	//    total /= timer->size();
	//    *avg_time = total;
	//};

	void startTimer(hipStream_t s)
	{
		hipEventCreate(&gen_start); hipEventCreate(&gen_stop);
		hipEventRecord(gen_start, s);
	};
	void stopTimer(hipStream_t s)
	{
		hipEventRecord(gen_stop, s); hipEventSynchronize(gen_stop);
	};
	void printTimer()
	{
		hipEventElapsedTime(&gen_elapsed, gen_start, gen_stop);
		printf("%.4f ms", gen_elapsed);
	};
	void destroyTimer(hipStream_t s)
	{
		hipEventDestroy(gen_start);
		hipEventDestroy(gen_stop);
	};


	//#define STORE_TIMER(t)   float elapsedTime2; hipEventElapsedTime(&elapsedTime2, start, stop); t.push_back(elapsedTime2);

	/////////// functions added for CUDA 5.0 compability ///////////////

	inline int cudaDeviceInit(int ARGC, char **ARGV)
	{
		int cuDevice = 0;
		int deviceCount = 0;
		hipError_t err = hipInit(0);

		if (hipSuccess == err)
		{
			checkCudaErrors(hipGetDeviceCount(&deviceCount));
		}

		if (deviceCount == 0)
		{
			fprintf(stderr, "cudaDeviceInit error: no devices supporting CUDA\n");
			exit(EXIT_FAILURE);
		}

		int dev = 0;
		dev = getCmdLineArgumentInt(ARGC, (const char **) ARGV, "device=");

		if (dev < 0)
		{
			dev = 0;
		}

		if (dev > deviceCount-1)
		{
			fprintf(stderr, "\n");
			fprintf(stderr, ">> %d CUDA capable GPU device(s) detected. <<\n", deviceCount);
			fprintf(stderr, ">> cudaDeviceInit (-device=%d) is not a valid GPU device. <<\n", dev);
			fprintf(stderr, "\n");
			return -dev;
		}

		checkCudaErrors(hipDeviceGet(&cuDevice, dev));
		char name[100];
		hipDeviceGetName(name, 100, cuDevice);

		if (checkCmdLineFlag(ARGC, (const char **) ARGV, "quiet") == false)
		{
			printf("> Using CUDA Device [%d]: %s\n", dev, name);
		}

		return dev;
	}

		// This function returns the best GPU based on performance
	inline int getMaxGflopsDeviceId()
	{
		hipDevice_t current_device = 0, max_perf_device = 0;
		int device_count     = 0, sm_per_multiproc = 0;
		int max_compute_perf = 0, best_SM_arch     = 0;
		int major = 0, minor = 0, multiProcessorCount, clockRate;

		hipInit(0);
		checkCudaErrors(hipGetDeviceCount(&device_count));

		// Find the best major SM Architecture GPU device
		while (current_device < device_count)
		{
			checkCudaErrors(hipDeviceComputeCapability(&major, &minor, current_device));

			if (major > 0 && major < 9999)
			{
				best_SM_arch = MAX(best_SM_arch, major);
			}

			current_device++;
		}

		// Find the best CUDA capable GPU device
		current_device = 0;

		while (current_device < device_count)
		{
			checkCudaErrors(hipDeviceGetAttribute(&multiProcessorCount,
												 hipDeviceAttributeMultiprocessorCount,
												 current_device));
			checkCudaErrors(hipDeviceGetAttribute(&clockRate,
												 hipDeviceAttributeClockRate,
												 current_device));
			checkCudaErrors(hipDeviceComputeCapability(&major, &minor, current_device));

			if (major == 9999 && minor == 9999)
			{
				sm_per_multiproc = 1;
			}
			else
			{
				sm_per_multiproc = _ConvertSMVer2Cores(major, minor);
			}

			int compute_perf  = multiProcessorCount * sm_per_multiproc * clockRate;

			if (compute_perf  > max_compute_perf)
			{
				// If we find GPU with SM major > 2, search only these
				if (best_SM_arch > 2)
				{
					// If our device==dest_SM_arch, choose this, or else pass
					if (major == best_SM_arch)
					{
						max_compute_perf  = compute_perf;
						max_perf_device   = current_device;
					}
				}
				else
				{
					max_compute_perf  = compute_perf;
					max_perf_device   = current_device;
				}
			}

			++current_device;
		}

		return max_perf_device;
	}


} // extern "C"

#endif
