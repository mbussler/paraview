#include "hip/hip_runtime.h"
// System includes
#include "cudaRidge.h"
#include <vector>
#include <set>
#include <thrust/device_ptr.h>

#include "mctable.h"
#include "kernels.cu"

#define CUDA_OUTPUT_STEPS 1
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

typedef double   vec3[3];
typedef  vec3    mat3[3];

bool cudaRidgeData( uint dim[3], 
                    const double* data, 
                    const double* grad, 
                    const double* hesse, 
                    double** evals_out, 
                    uint* numVertices_out, 
                    double** vertices_out, 
                    double** pointData_out, 
                    double featureThreshold, 
                    double origin[3], 
                    double spacing[3], 
                    unsigned int stencilRange, 
                    bool valley) 
{
#ifdef CUDA_OUTPUT_STEPS
    std::cout << "    CUDA: Starting cuda method..." << std::endl;
#endif

    int deviceCount = 0;
    gpuErrchk(hipGetDeviceCount(&deviceCount));
    std::cout << "    CUDA: number of devices is " << deviceCount << std::endl;

    if (hipSetDevice(0) != hipSuccess) {
    std::cout << "    CUDA: Could not set cuda device." << std::endl;
    }
    else {
#ifdef CUDA_OUTPUT_STEPS
    std::cout << "    CUDA: Cuda device set." << std::endl;
#endif
    }
    size_t numPointsTotal=dim[0]*dim[1]*dim[2];
    size_t sgTotal = 3*numPointsTotal*sizeof(double);
    size_t shesseTotal = 3*3*numPointsTotal*sizeof(double);
    size_t seValTotal = 3*numPointsTotal*sizeof(double);
    size_t seVectTotal = 3*numPointsTotal*sizeof(double);
    size_t slargeEVTotal = 3*numPointsTotal*sizeof(double);
    size_t sdTotal = numPointsTotal*sizeof(double);
    size_t sbitmasksTotal = 2*numPointsTotal*sizeof(uint);
    size_t sdim = 3*sizeof(uint);
    size_t snVT = 256*sizeof(uint);
    size_t stT = 256*16*sizeof(uint);
    uint *dim_d;

    size_t memoryRequired = sdTotal + sgTotal + seValTotal + seVectTotal + slargeEVTotal + shesseTotal + sbitmasksTotal + sdim + snVT + stT;
    size_t memoryFree, memoryTotal;
    hipMemGetInfo(&memoryFree,&memoryTotal);

    size_t memoryKeepFree = 250l*1024l*1024l;
    size_t memoryMin = 1100*1024*1024;
    size_t memoryToUse = min(memoryFree - memoryKeepFree, memoryMin );
    size_t numParts = 1;
    size_t maxNumSlicesInMemory = dim[2];

    std::cout << "Memory on device: " << memoryFree << " byte free (" << memoryTotal << " byte total), " << memoryRequired << " byte required." << std::endl;
    std::cout << "Using at most " << memoryToUse << " bytes." << std::endl;

    unsigned int overlap = 4*stencilRange;

    if (memoryToUse < memoryRequired)
    {
        //calculate memory required for overlapping slice of data for the blocks
        size_t numPointsSlice=dim[0]*dim[1];
        size_t sgSlice = 3*numPointsSlice*sizeof(double);
        size_t shesseSlice = 3*3*numPointsSlice*sizeof(double);
        size_t seValSlice = 3*numPointsSlice*sizeof(double);
        size_t seVectSlice = 3*numPointsSlice*sizeof(double);
        size_t slargeEVSlice = 3*numPointsSlice*sizeof(double);
        size_t sdSlice = numPointsSlice*sizeof(double);
        size_t sbitmasksSlice = 2*numPointsSlice*sizeof(uint);
        size_t memoryRequiredSlice = sdSlice + sgSlice + seValSlice + seVectSlice + slargeEVSlice + shesseSlice + sbitmasksSlice + sdim + snVT + stT;

        //add to required memory per part and test, if data needs to be split into more parts
        maxNumSlicesInMemory = memoryToUse / memoryRequiredSlice;
        numParts = (dim[2] - 1) / maxNumSlicesInMemory + 1;
        numParts += numParts*overlap/maxNumSlicesInMemory;
        while (dim[2] - (numParts-1) * (maxNumSlicesInMemory - overlap) > maxNumSlicesInMemory)
        {
            ++numParts;
        }
//        numParts = (memoryRequired - 1) / (memoryRequiredSlice * (maxNumSlicesInMemory - 1)) + 1; // maxNumSlicesInMemory - 1 to account for overlapping slice
        size_t memoryRequiredPart = memoryRequiredSlice * (maxNumSlicesInMemory - 1);
//        if (memoryToUse < memoryRequiredPart)
//        {
//            ++numParts;
//        }

        std::cout << "Data set needs to be split into " << numParts << " parts of " << memoryRequiredPart << " bytes + " << " bytes kept free." << std::endl;

//        hipDeviceReset();
//        return false;
    }

    // double minEVal = -0.1;
    double *hesse_d;
    double *g_d, *d_d, *eVect_d, *evals_d, *largeEV_d;
    uint *bitmasks_d, *nVT_d, *tT_d;

    size_t numPointsPart=dim[0]*dim[1]*maxNumSlicesInMemory;
    size_t sgPart = 3*numPointsPart*sizeof(double);
    size_t shessePart = 3*3*numPointsPart*sizeof(double);
    size_t seValPart = 3*numPointsPart*sizeof(double);
    size_t seVectPart = 3*numPointsPart*sizeof(double);
    size_t slargeEVPart = 3*numPointsPart*sizeof(double);
    size_t sdPart = numPointsPart*sizeof(double);
    size_t sbitmasksPart = 2*numPointsPart*sizeof(uint);

    std::vector< std::vector<double> > verticesPart(numParts);
    std::vector< std::vector<double> > evalsPart(numParts);
    std::vector< std::vector<double> > pointDataPart(numParts);

    size_t numVerticesTotal = 0;

    gpuErrchk(hipMalloc((void**)&d_d, sdPart));
    gpuErrchk(hipMalloc((void**)&g_d, sgPart));
    gpuErrchk(hipMalloc((void**)&evals_d, seValPart));
    gpuErrchk(hipMalloc((void**)&eVect_d, seVectPart));
    gpuErrchk(hipMalloc((void**)&largeEV_d, slargeEVPart));
    gpuErrchk(hipMalloc((void**)&hesse_d, shessePart));
    gpuErrchk(hipMalloc((void**)&bitmasks_d, sbitmasksPart));
    gpuErrchk(hipMalloc((void**)&dim_d, sdim));
    gpuErrchk(hipMalloc((void**)&nVT_d, snVT));
    gpuErrchk(hipMalloc((void**)&tT_d, stT));

#ifdef CUDA_OUTPUT_STEPS
    std::cout << "    CUDA: Memory allocated." << std::endl;
#endif

    for (size_t i = 0; i < numParts; i++)
    {
//        verticesPart[i] = NULL;
//        evalsPart[i] = NULL;
//        pointDataPart[i] = NULL;
        hipMemGetInfo(&memoryFree,&memoryTotal);
        std::cout << "    CUDA: Available device memory: " << memoryFree << " bytes. " << std::endl;

        size_t sliceOffset = i * (maxNumSlicesInMemory - overlap);
        //size_t sliceOffset = i * maxNumSlicesInMemory;
        size_t offsetIndex = sliceOffset * dim[0] * dim[1];
        size_t numSlices = i < numParts - 1 ? maxNumSlicesInMemory : dim[2] - sliceOffset;
        size_t numPoints = dim[0]*dim[1]*numSlices;
        //numActualPointsPart[i] = numPoints;
        size_t sd = numPoints*sizeof(double);
        //size_t sbitmasks = 2*numPoints*sizeof(uint);
        std::cout << "    CUDA: slice offset: " << sliceOffset << " number of slices " << numSlices << " number of points " << numPoints << std::endl;

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //uint *bitmasks_h = new uint[2*numPoints];
        gpuErrchk(hipMemcpy(d_d, data + offsetIndex, sd, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(nVT_d, numVertsTable, snVT, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(tT_d, triTable, stT, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(dim_d, dim, sdim, hipMemcpyHostToDevice));

        gpuErrchk(hipMemset(g_d, 0, sgPart));
//        double *d_debug = new double[numPoints];
//        gpuErrchk(hipMemcpy(d_d, d + offsetIndex, sd, hipMemcpyHostToDevice));
//        gpuErrchk(hipMemcpy(nVT_d, numVertsTable, snVT, hipMemcpyHostToDevice));
//        gpuErrchk(hipMemcpy(tT_d, triTable, stT, hipMemcpyHostToDevice));
//        gpuErrchk(hipMemcpy(dim_d, dim, sdim, hipMemcpyHostToDevice));

    #ifdef CUDA_OUTPUT_STEPS
        std::cout << "    CUDA: Memory copied." << std::endl;
    #endif
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        uint blockSize = 128;
        int nBlocks = numPoints/blockSize + (numPoints%blockSize == 0?0:1);

        if( grad ) {
            gpuErrchk(hipMemcpy(g_d, grad + 3*offsetIndex, 3*sd, hipMemcpyHostToDevice));
        } else {
            //calculate gradients if not given
            processPointGradients <<< nBlocks, blockSize >>> (numPoints, d_d, g_d, dim_d, stencilRange, valley);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
        }

        //use those to calculate hessian matrix
        if( hesse ) {
            gpuErrchk(hipMemcpy(hesse_d, hesse + 9*offsetIndex, 9*sd, hipMemcpyHostToDevice));
        } else {
            processPointHesse <<< nBlocks, blockSize >>> (numPoints, g_d, hesse_d, dim_d, stencilRange);
            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
        }
        //use those to calculate eigenvalues
        processPointEigenVals <<< nBlocks, blockSize >>> (numPoints, hesse_d, evals_d, dim_d);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //use those to calculate eigenvectors
        processPointEigenVect <<< nBlocks, blockSize >>> (numPoints, hesse_d, evals_d, eVect_d, dim_d);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //use those to calculate an array of vectors giving the general orientation of eigenvectors per cube
        largestEigenVect <<< nBlocks, blockSize >>> (numPoints, eVect_d, largeEV_d, dim_d);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //generate bitmasks, arrays for prefix scan
        processEigenData <<< nBlocks, blockSize >>> ( numPoints, dim_d, 
                                                      evals_d, eVect_d, g_d, 
                                                      largeEV_d, bitmasks_d, 
                                                      nVT_d, d_d, featureThreshold);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //gpuErrchk(hipMemcpy(bitmasks_h, bitmasks_d, sbitmasks, hipMemcpyDeviceToHost));

        //uint *pfA_h = new uint[2*numPoints];
        uint *pfA_d;
        size_t pfAB = 2*numPoints*sizeof(uint);
        gpuErrchk(hipMalloc( (void**)&pfA_d, pfAB));

        thrust::device_ptr<uint> prefixArray = thrust::device_pointer_cast(pfA_d);
        thrust::device_ptr<uint> bitmaskArray = thrust::device_pointer_cast(bitmasks_d);

        int addOne = 0;
        int addTris = 0;
        if (bitmaskArray[2*numPoints-1]) {
            addOne = 1;                    //doing exclusive scan, if last element is set, we need one more space
            addTris = bitmaskArray[2*numPoints-1];  //and the tris of course
        }

//        thrust::exclusive_scan(&bitmasks_h[numPoints], bitmasks_h+2*numPoints, &pfA_h[numPoints]);  //index of next cubes
//        thrust::exclusive_scan(&bitmasks_h[0], bitmasks_h+numPoints, &pfA_h[0]);  //index of next cubes
        thrust::exclusive_scan(bitmaskArray + numPoints, bitmaskArray + 2*numPoints, prefixArray + numPoints);  //index of next cubes
        thrust::exclusive_scan(bitmaskArray, bitmaskArray + numPoints, prefixArray);  //index of next cubes

        uint numIsoSurfaceCubes=prefixArray[2*numPoints-1]+addOne;
        uint numVertices=prefixArray[numPoints-1]+addTris;

        if (numIsoSurfaceCubes == 0)
        {
            //numVerticesPart[i] = 0;
            //evalsPart[i] = new double[numPoints*3];
            evalsPart[i].resize(numPoints*3);
            gpuErrchk(hipMemcpy(evalsPart[i].data(), evals_d, numPoints*3*sizeof(double), hipMemcpyDeviceToHost));
            continue;
        }
        numVerticesTotal += numVertices;


    #ifdef CUDA_OUTPUT_STEPS
        cout << "    CUDA: numIsoSurfaceCubes: " << numIsoSurfaceCubes << endl;
        cout << "    CUDA: numVertices: " << numVertices << endl;
    #endif

        size_t siA, svA, sgA, sizePointData;
        uint *iA_d;
        double *vA_d;//, *vA_h = new double[numVertices*3];
        verticesPart[i].resize(numVertices*3);
        double *gA_d;
        //double *evals_h = new double[numPoints*3];
        evalsPart[i].resize(numPoints*3);
        double *pointData_d;//, *pointData_h = new double[numVertices];
        pointDataPart[i].resize(numVertices);
        siA = numIsoSurfaceCubes*sizeof(uint);
        svA = numVertices*3*sizeof(double);
        sgA = numVertices*3*sizeof(double);
        sizePointData = numVertices*sizeof(double);

        gpuErrchk(hipMalloc( (void**)&iA_d,  siA));
        gpuErrchk(hipMalloc( (void**)&vA_d,  svA));
        gpuErrchk(hipMalloc( (void**)&gA_d,  sgA));
        gpuErrchk(hipMalloc( (void**)&pointData_d, sizePointData));

        //gpuErrchk(hipMemcpy( pfA_d, pfA_h, pfAB, hipMemcpyHostToDevice));   //copy prefix array for second pass
        getIsoIndices <<< nBlocks, blockSize >>> (numPoints, iA_d, bitmasks_d, pfA_d);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
//        std::vector<unsigned int> cellIndices(numIsoSurfaceCubes);
//        gpuErrchk(hipMemcpy(cellIndices.data(), iA_d, siA, hipMemcpyDeviceToHost));

//        std::set<unsigned int> nodeIndicesTmp;
//        for (std::vector<unsigned int>::iterator it = cellIndices.begin(); it != cellIndices.end(); it++)
//        {
//            nodeIndicesTmp.insert(*it);
//            nodeIndicesTmp.insert(*it + 1);
//            nodeIndicesTmp.insert(*it + dim[0]);
//            nodeIndicesTmp.insert(*it + dim[0] + 1);
//            nodeIndicesTmp.insert(*it + dim[1]*dim[0]);
//            nodeIndicesTmp.insert(*it + dim[1]*dim[0] + 1);
//            nodeIndicesTmp.insert(*it + dim[1]*dim[0] + dim[0]);
//            nodeIndicesTmp.insert(*it + dim[1]*dim[0] + dim[0] + 1);
//        }

        nBlocks = numIsoSurfaceCubes/blockSize + (numIsoSurfaceCubes%blockSize == 0?0:1);
        double3 originTmp;
        originTmp.x = origin[0];
        originTmp.y = origin[1];
        originTmp.z = origin[2] + sliceOffset;
        double3 spacingTmp;
        spacingTmp.x = spacing[0];
        spacingTmp.y = spacing[1];
        spacingTmp.z = spacing[2];
        processRidgeCubesData <<< nBlocks, blockSize >>> (numIsoSurfaceCubes, dim_d, iA_d, pfA_d, eVect_d, g_d, largeEV_d, vA_d, gA_d, nVT_d, tT_d, bitmasks_d, d_d, pointData_d, originTmp, spacingTmp);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        //gpuErrchk(hipMemcpy(vA_h, vA_d, svA, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(verticesPart[i].data(), vA_d, svA, hipMemcpyDeviceToHost));
        //gpuErrchk(hipMemcpy(evals_h, evals_d, sgA, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(evalsPart[i].data(), evals_d, evalsPart[i].size()*sizeof(double), hipMemcpyDeviceToHost));
        //gpuErrchk(hipMemcpy(pointData_h, pointData_d, sizePointData, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(pointDataPart[i].data(), pointData_d, sizePointData, hipMemcpyDeviceToHost));

        gpuErrchk(hipFree(iA_d));
        gpuErrchk(hipFree(vA_d));
        gpuErrchk(hipFree(gA_d));
        gpuErrchk(hipFree(pfA_d));
        gpuErrchk(hipFree(pointData_d));
        cout << "vram released" << endl;

        //delete [] bitmasks_h;
        //delete [] pfA_h;
        cout << "host mem released" << endl;

        //numVerticesPart[i] = numVertices;
        //verticesPart[i] = vA_h;
        //evalsPart[i] = evals_h;
        //pointDataPart[i] = pointData_h;
//        *numVertices_out = numVertices;
//        *vertices_out = vA_h;
//        *gradients_out = gA_h;
//        *pointData_out = pointData_h;
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
    }
    gpuErrchk(hipFree(hesse_d));
    gpuErrchk(hipFree(evals_d));

    gpuErrchk(hipFree(dim_d));
    gpuErrchk(hipFree(d_d));
    gpuErrchk(hipFree(g_d));
    gpuErrchk(hipFree(eVect_d));
    gpuErrchk(hipFree(largeEV_d));


    double *vertices = new double[numVerticesTotal*3];
    double *evals = new double[numPointsTotal*3];
    double *pointData = new double[numVerticesTotal];

    if (!(vertices && evals && pointData))
    {
        return false;
    }

    size_t indexOffset = 0;
    size_t indexOffsetEval = 0;
    //copy everything together
    for (size_t i = 0; i < numParts; i++)
    {
        //copy
        if (!verticesPart[i].empty())
        {
            memcpy(vertices + indexOffset*3, verticesPart[i].data(), verticesPart[i].size()*sizeof(double));
            memcpy(pointData + indexOffset, pointDataPart[i].data(), pointDataPart[i].size()*sizeof(double));
            indexOffset += pointDataPart[i].size();
        }

        //account for overlapping slice
        size_t numEvalsSlice = 3*(dim[0]*dim[1]);
        size_t sliceOffset = i == 0 ? 0 : overlap/2;
        size_t numEvalsToCopy = (i > 0 && i < numParts - 1) ? evalsPart[i].size() - overlap*numEvalsSlice : evalsPart[i].size() - (overlap/2)*numEvalsSlice;
        if (numParts == 1) numEvalsToCopy = evalsPart[i].size();
        memcpy(evals + indexOffsetEval, evalsPart[i].data() + sliceOffset*numEvalsSlice, numEvalsToCopy*sizeof(double));
        indexOffsetEval += numEvalsToCopy;
    }

//    size_t indexOffset = 0;
//    size_t indexOffsetEval = 0;
//    //copy everything together
//    for (size_t i = 0; i < numParts; i++)
//    {
//        //copy
//        if (numVerticesPart[i] > 0)
//        {
//            memcpy(vertices + indexOffset*3, verticesPart[i], 3*numVerticesPart[i]*sizeof(double));
//            memcpy(pointData + indexOffset, pointDataPart[i], numVerticesPart[i]*sizeof(double));
//            indexOffset += numVerticesPart[i];
//            if (verticesPart[i]) delete[] verticesPart[i];
//            if (pointDataPart[i]) delete[] pointDataPart[i];
//        }

//        //account for overlapping slice
//        size_t numEvalsToCopy = i < numParts - 1 ? 3*(numActualPointsPart[i] - dim[0]*dim[1]) : 3*(numActualPointsPart[i]);
//        memcpy(evals + indexOffsetEval*3, evalsPart[i], numEvalsToCopy*sizeof(double));
//        indexOffsetEval += numEvalsToCopy;
//        //delete
//        if (evalsPart[i]) delete[] evalsPart[i];
//    }

    *numVertices_out = numVerticesTotal;
    *vertices_out = vertices;
    *evals_out = evals;
    *pointData_out = pointData;

    hipDeviceReset();

    return true;
}
